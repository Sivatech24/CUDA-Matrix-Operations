
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#define N 2 // 2x2 matrix

__global__ void fillRandomMatrix(int *matrix, unsigned long seed) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < N && col < N) {
        int idx = row * N + col;

        // Set up CURAND
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        // Generate a random number between 0 and 99
        matrix[idx] = hiprand(&state) % 100;
    }
}

__global__ void displayMatrix(int *matrix) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < N && col < N) {
        int idx = row * N + col;
        printf("%2d ", matrix[idx]);
        if (col == N - 1) printf("\n");
    }
}

int main() {
    int *d_matrix;

    hipMalloc(&d_matrix, sizeof(int) * N * N);

    dim3 threadsPerBlock(N, N);

    // Fill with random numbers
    fillRandomMatrix<<<1, threadsPerBlock>>>(d_matrix, time(NULL));
    hipDeviceSynchronize();

    std::cout << "Random 2x2 Matrix:\n";
    displayMatrix<<<1, threadsPerBlock>>>(d_matrix);
    hipDeviceSynchronize();

    hipFree(d_matrix);
    return 0;
}
