
#include <hip/hip_runtime.h>
#include <iostream>
#include <hiprand/hiprand_kernel.h>

#define N 16 // 16x16 matrix

__global__ void fillRandomMatrix(int *matrix, unsigned long seed) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < N && col < N) {
        int idx = row * N + col;

        // Initialize CURAND
        hiprandState state;
        hiprand_init(seed, idx, 0, &state);

        // Random int between 0 and 99
        matrix[idx] = hiprand(&state) % 100;
    }
}

__global__ void displayMatrix(int *matrix) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    if (row < N && col < N) {
        int idx = row * N + col;
        printf("%2d ", matrix[idx]);
        if (col == N - 1) printf("\n");
    }
}

int main() {
    int *d_matrix;

    // Allocate device memory
    hipMalloc(&d_matrix, sizeof(int) * N * N);

    // Launch kernel to fill matrix with random numbers
    dim3 threadsPerBlock(N, N);
    fillRandomMatrix<<<1, threadsPerBlock>>>(d_matrix, time(NULL));
    hipDeviceSynchronize();

    std::cout << "Random 16x16 Matrix:\n";
    displayMatrix<<<1, threadsPerBlock>>>(d_matrix);
    hipDeviceSynchronize();

    hipFree(d_matrix);
    return 0;
}
